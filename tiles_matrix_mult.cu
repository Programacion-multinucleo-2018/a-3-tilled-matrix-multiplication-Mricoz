#include "hip/hip_runtime.h"
/*
    Matrix Multiplication on GPU with tiles
*/

#include <iostream>
#include <cstdio>
#include <cstdlib>
#include <math.h>
#include <chrono>

#include "common.h"

#define MATRIX_SIZE 2000 // Matrix size
#define TILE_SIZE 8 // tiles size

// Fill the matrix
void fillMatrix(float * M){
    int size = MATRIX_SIZE * MATRIX_SIZE;
    for(int i = 0; i < size; i++){
        M[i] = (float)rand()/(RAND_MAX/ 10.0f);
    }
    return;
}

// Print the matrix
void printMatrix(float * M){
    int size = MATRIX_SIZE * MATRIX_SIZE;
    for(int x = 0; x < size; x++){
        if(x % MATRIX_SIZE == 0){
            printf("\n");
        }
        printf("%f ", M[x]);
    }
}

// Check results
int checkResult(float * hostRef, float * gpuRef){
    double epsilon = 0.5;
    bool match = 1;
    int size = MATRIX_SIZE * MATRIX_SIZE;
    for (int i = 0; i < size; i++){
        if (abs(hostRef[i] - gpuRef[i]) > epsilon){
            match = 0;
            printf("host %f gpu %f dif %f\n", hostRef[i], gpuRef[i],hostRef[i] - gpuRef[i]);
            break;
        }
    }
    return match;
}

// Multiply in CPU
void multiplyMatrixCPU(float * C, float * A, float * B){
    for(int y = 0; y < MATRIX_SIZE; y++){
        for(int z = 0; z < MATRIX_SIZE; z++){
            for(int x = 0; x < MATRIX_SIZE; x++){
                C[y * MATRIX_SIZE + z] += A[x + y * MATRIX_SIZE] * B[x * MATRIX_SIZE + z];
            }
        }
    }
}

// Multiply in GPU no tiles
__global__ void multiplyMatrixGPU(float * A, float * B, float * C){
    unsigned int ix = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned int iy = threadIdx.y + blockIdx.y * blockDim.y;
    float sum = 0;
    if (ix < MATRIX_SIZE && iy < MATRIX_SIZE){
        for(int j = 0; j < MATRIX_SIZE; j++){
            sum += A[ix * MATRIX_SIZE + j] * B[j * MATRIX_SIZE + iy];
        }
        C[ix * MATRIX_SIZE + iy] = sum;
    }
}

// Multiplicar GPU con tiles
__global__ void multiplyMatrixGPUTiles(float * A, float * B, float * C){
    float sum = 0;
    unsigned int ix = threadIdx.x + TILE_SIZE * blockIdx.x;
    unsigned int iy = threadIdx.y + TILE_SIZE * blockIdx.y;
    unsigned int x = threadIdx.x;
    unsigned int y = threadIdx.y;
    // Shared variables
    __shared__ float sharedA[TILE_SIZE][TILE_SIZE];
    __shared__ float sharedB[TILE_SIZE][TILE_SIZE];

    sharedA[y][x] = 0.0; // inicia 0
    sharedB[y][x] = 0.0; // inicia 0

    __syncthreads();

    // i-- para seguir con los 0
    for (int i = (TILE_SIZE + MATRIX_SIZE - 1) / TILE_SIZE; i >= 0; i--) {
        if ((i * TILE_SIZE + x ) < MATRIX_SIZE && iy < MATRIX_SIZE) {
            sharedA[y][x] = A[iy * MATRIX_SIZE + i * TILE_SIZE + x];
        }
        if ((i * TILE_SIZE + y) < MATRIX_SIZE && ix < MATRIX_SIZE){
            sharedB[y][x] = B[(i * TILE_SIZE + y) * MATRIX_SIZE + ix];
        }

        __syncthreads(); // sync

        for (int j = 0; j < TILE_SIZE; j++){
             sum += sharedA[y][j] * sharedB[j][x];
        }
        __syncthreads(); // sync
    }

    if (ix < MATRIX_SIZE && iy < MATRIX_SIZE){
      C[iy * MATRIX_SIZE + ix] = sum;
    }
}

int main(int argc, char **argv){
    printf("%s Starting...\n", argv[0]);

    // set up device
    int dev = 0;
    hipDeviceProp_t deviceProp;
    SAFE_CALL(hipGetDeviceProperties(&deviceProp, dev), "Error device prop");
    printf("Using Device %d: %s\n", dev, deviceProp.name);
    SAFE_CALL(hipSetDevice(dev), "Error setting device");

    // set up data size of matrix
    int nx = MATRIX_SIZE;
    int ny = MATRIX_SIZE;

    int nxy = nx * ny;
    int nBytes = nxy * sizeof(float);
    printf("Matrix size: x %d y %d\n", nx, ny);
    std::cout << "Tile size: " << TILE_SIZE << "x" << TILE_SIZE <<'\n';
    std::cout << '\n';


    // malloc host memory
    float *h_m1, *h_m2, *hostRef, *gpuRef, *gpuRefTiles;
    h_m1 = (float *)malloc(nBytes);
    h_m2 = (float *)malloc(nBytes);
    hostRef = (float *)malloc(nBytes);
    gpuRef = (float *)malloc(nBytes);
    gpuRefTiles = (float *)malloc(nBytes);

    fillMatrix(h_m1); // initialize data at host side
    fillMatrix(h_m2); // initialize data at host side

    memset(hostRef, 0, nBytes);
    memset(gpuRefTiles, 0, nBytes);
    memset(gpuRef, 0, nBytes);

    ////////// CPU for reference //////////
    auto start_cpu =  std::chrono::high_resolution_clock::now();
    multiplyMatrixCPU(hostRef, h_m1, h_m2);
    auto end_cpu =  std::chrono::high_resolution_clock::now();
    std::chrono::duration<float, std::milli> duration_ms = end_cpu - start_cpu;

    printf("multiplyMatrixCPU elapsed %f ms\n\n", duration_ms.count());

    // malloc device global memory
    float *d_MatA, *d_MatB, *d_MatC;
    SAFE_CALL(hipMalloc((void **)&d_MatA, nBytes), "Error allocating d_MatA");
    SAFE_CALL(hipMalloc((void **)&d_MatB, nBytes), "Error allocating d_MatB");
    SAFE_CALL(hipMalloc((void **)&d_MatC, nBytes), "Error allocating d_MatC");

    // transfer data from host to device
    SAFE_CALL(hipMemcpy(d_MatA, h_m1, nBytes, hipMemcpyHostToDevice), "Error copying d_MatA");
    SAFE_CALL(hipMemcpy(d_MatB, h_m2, nBytes, hipMemcpyHostToDevice), "Error copying d_MatB");

    // invoke kernel at host side
    int dimx = TILE_SIZE;
    int dimy = TILE_SIZE;
    dim3 block(dimx, dimy);
    dim3 grid((nx + block.x - 1) / block.x, (ny + block.y - 1) / block.y);

    ////////// GPU no tiles //////////
    start_cpu =  std::chrono::high_resolution_clock::now();
    multiplyMatrixGPU<<<grid, block>>>(d_MatA, d_MatB, d_MatC);
    SAFE_CALL(hipDeviceSynchronize(), "Error executing kernel");
    end_cpu =  std::chrono::high_resolution_clock::now();
    duration_ms = end_cpu - start_cpu;

    printf("multiplyMatrixGPU <<<(%d,%d), (%d,%d)>>> elapsed %f ms\n", grid.x, grid.y, block.x, block.y, duration_ms.count());

    // SAFE_CALL kernel error
    SAFE_CALL(hipGetLastError(), "Error with last error");

    // copy kernel result back to host side
    SAFE_CALL(hipMemcpy(gpuRef, d_MatC, nBytes, hipMemcpyDeviceToHost), "Error copying d_MatC");
    if(checkResult(hostRef, gpuRef))
      printf("MATCH\n\n");
    else
      printf("NO MATCH\n\n");

    ////////// GPU with tiles //////////
    start_cpu =  std::chrono::high_resolution_clock::now();
    multiplyMatrixGPUTiles<<<grid, block>>>(d_MatA, d_MatB, d_MatC);
    SAFE_CALL(hipDeviceSynchronize(), "Error executing kernel");
    end_cpu =  std::chrono::high_resolution_clock::now();
    duration_ms = end_cpu - start_cpu;

    printf("multiplyMatrixGPUTiles <<<(%d,%d), (%d,%d)>>> elapsed %f ms\n", grid.x,grid.y,block.x, block.y, duration_ms.count());

    // SAFE_CALL kernel error
    SAFE_CALL(hipGetLastError(), "Error with last error");

    // copy kernel result back to host side
    SAFE_CALL(hipMemcpy(gpuRefTiles, d_MatC, nBytes, hipMemcpyDeviceToHost), "Error copying d_MatC");
    if(checkResult(hostRef, gpuRefTiles))
      printf("MATCH\n\n");
    else
      printf("NO MATCH\n\n");

    // free device global memory
    SAFE_CALL(hipFree(d_MatA), "Error freeing memory");
    SAFE_CALL(hipFree(d_MatB), "Error freeing memory");
    SAFE_CALL(hipFree(d_MatC), "Error freeing memory");

    // free host memory
    free(h_m1);
    free(h_m2);
    free(hostRef);
    free(gpuRef);

    // reset device
    SAFE_CALL(hipDeviceReset(), "Error reseting");

    return (0);
}
